/*
 * Copyright 2021 NVIDIA Corporation. All rights reserved
 *
 * Sample to show how to correlate CUDA APIs with the corresponding GPU
 * activities using the correlation-id field in the activity records.
 *
 */
#include <hip/hip_runtime.h>

#include <cupti.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
using namespace std;
#define DRIVER_API_CALL(apiFuncCall)                                            \
    do                                                                          \
    {                                                                           \
        hipError_t _status = apiFuncCall;                                         \
        if (_status != hipSuccess)                                            \
        {                                                                       \
            const char* errstr;                                                 \
            hipDrvGetErrorString(_status, &errstr);                                 \
            fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",\
                     __FILE__, __LINE__, #apiFuncCall, errstr);                 \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    } while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                               \
    do                                                                              \
    {                                                                               \
        hipError_t _status = apiFuncCall;                                          \
        if (_status != hipSuccess)                                                 \
        {                                                                           \
            fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",    \
                    __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status)); \
            exit(EXIT_FAILURE);                                                     \
        }                                                                           \
    } while (0)

#define COMPUTE_N 50000

extern void initTrace(void);
extern void finiTrace(void);

// Kernels
__global__ void
VecAdd(const int* A, const int* B, int* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

__global__ void
VecSub(const int* A, const int* B, int* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
        C[i] = A[i] - B[i];
    }
}

static void
do_pass(hipStream_t stream)
{
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;
    size_t size = COMPUTE_N * sizeof(int);
    int threadsPerBlock = 256;
    int blocksPerGrid = 0;

    // Allocate input vectors h_A and h_B in host memory
    // don't bother to initialize
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);
    if (!h_A || !h_B || !h_C)
    {
        printf("Error: out of memory\n");
        exit(EXIT_FAILURE);
    }

    // Allocate vectors in device memory
    RUNTIME_API_CALL(hipMalloc((void**)&d_A, size));
    RUNTIME_API_CALL(hipMalloc((void**)&d_B, size));
    RUNTIME_API_CALL(hipMalloc((void**)&d_C, size));

    RUNTIME_API_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream));
    RUNTIME_API_CALL(hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream));

    blocksPerGrid = (COMPUTE_N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, COMPUTE_N);
    VecSub<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, COMPUTE_N);

    RUNTIME_API_CALL(hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream));

    if (stream == 0)
    {
        RUNTIME_API_CALL(hipDeviceSynchronize());
    }
    else
    {
        RUNTIME_API_CALL(hipStreamSynchronize(stream));
    }
    free(h_A);
    free(h_B);
    free(h_C);
    RUNTIME_API_CALL(hipFree(d_A));
    RUNTIME_API_CALL(hipFree(d_B));
    RUNTIME_API_CALL(hipFree(d_C));
}

int main(int argc, char* argv[])
{
    hipDevice_t device;
    char deviceName[256];

    // initialize the activity trace
    initTrace();

    DRIVER_API_CALL(hipInit(0));
    DRIVER_API_CALL(hipDeviceGet(&device, 0));
    DRIVER_API_CALL(hipDeviceGetName(deviceName, 256, device));
    printf("Device Name: %s\n", deviceName);
    RUNTIME_API_CALL(hipSetDevice(0));

    // do pass default stream
    do_pass(0);

    // do pass with user stream
    hipStream_t stream0;
    RUNTIME_API_CALL(hipStreamCreate(&stream0));
    do_pass(stream0);
    RUNTIME_API_CALL(hipDeviceSynchronize());

    // Flush CUPTI buffers.
    finiTrace();
    exit(EXIT_SUCCESS);
}
